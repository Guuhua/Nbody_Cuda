#include "hip/hip_runtime.h"
/**
 * Nbody Cuda
 * @author Juntao Chen
 */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define BlockWidth 16
#define BlockSize 256
#define SOFTENING 1e-9f

typedef struct{ float x, y, z, vx, vy, vz; } Body;

// 初始化值为
void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

__global__ void bodyForce(Body *p, float dt, int n) {

  // 线程要处理的行号
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  // 判断边界，行号要小于n
  if (row < n) {
    
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
    
    // 循环网格维度x次
    for (int j = 0; j < gridDim.x; ++j) {
      
      // 定义共享变量存取block需要的body值
      __shared__ Body locN[BlockWidth];
      
      // 从全局内存中取出值
      Body loc1 = p[j * blockDim.x + threadIdx.x];
      
      // 放入共享内存中
      locN[threadIdx.x].x = loc1.x;
      locN[threadIdx.x].y = loc1.y;
      locN[threadIdx.x].z = loc1.z;
      
      // 确保所有的值都被取出才能进行下一步计算，避免因时差导致取值错误
      __syncthreads();
      
      for (int k = 0; k < BlockWidth; ++k) {
        float dx = locN[k].x - p[row].x;
        float dy = locN[k].y - p[row].y;
        float dz = locN[k].z - p[row].z;
        float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
        float invDist = sqrtf(distSqr);
        float invDist3 = invDist * invDist * invDist;

        Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
      }
      __syncthreads();
    }
    p[row].vx += dt*Fx; 
    p[row].vy += dt*Fy; 
    p[row].vz += dt*Fz;
  } 
}

int main(const int argc, const char** argv) {

  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations
  
  // 声明变量，在主机中存放值
  int bytes = nBodies * sizeof(Body);
  float *buf = (float *)malloc(bytes);
  Body *p = (Body*)buf;
  // 声明变量，在device存放值
  float *devBuf;
  hipMalloc(&devBuf, bytes);
  Body *devp = (Body*)devBuf;

  // 初始化变量
  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  // 计算得到块的个数
  int nBlocks = (nBodies + BlockSize -1) / BlockSize;

  for (int iter = 0; iter < nIters; iter++) {
    // 将值存入设备中    
    hipMemcpy(devp, buf, bytes, hipMemcpyHostToDevice);
    // 并行计算
    bodyForce<<<nBlocks, BLOCK_SIZE>>>(devp, dt, nBodies);
    // 将值从设备中取出来
    cudaMencpy(buf, devp, bytes, hipMemcpyDeviceToHost);
    for (int i = 0 ; i < nBodies; i++) { // integrate position
      p[i].x += p[i].vx*dt;
      p[i].y += p[i].vy*dt;
      p[i].z += p[i].vz*dt;
    }  
  }
  // 释放变量空间
  free(buf);
  cudafree(devBuf);
}